
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <hipblas.h>
#include <iostream>
#include <hiprand/hiprand.h>
#include <hip/hip_fp16.h>
#include <chrono>

void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
     // Create a pseudo-random number generator
     hiprandGenerator_t prng;
     hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

     // Set the seed for the random number generator using the system clock
     hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

     // Fill the array with random numbers on the device
     /* curandGenerateUniform(prng, A, nr_rows_A * nr_cols_A); */
}

void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
     int lda=m,ldb=k,ldc=m;

     float *alpha = new float(3.14515f);
     float *beta = new float(0.0f);


     // Create a handle for CUBLAS
     hipblasHandle_t handle;
     hipblasCreate(&handle);

     std::chrono::time_point<std::chrono::system_clock> start, end;

     // Do the actual multiplication
     start = std::chrono::system_clock::now();
     for (size_t i = 0; i < 1; ++i) {
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha,
            A, lda, B, ldb, beta, C, ldc);
        hipStreamSynchronize(0);
     }
            std::cerr << "COS\n";
     end = std::chrono::system_clock::now();

     std::chrono::duration<double> elapsed_seconds = end-start;
     std::time_t end_time = std::chrono::system_clock::to_time_t(end);
     std::cout << "finished computation at " << std::ctime(&end_time)
               << "elapsed time: " << elapsed_seconds.count() << "s\n";

     // Destroy the handle
     hipblasDestroy(handle);
}

int main() {
     // Allocate 3 arrays on CPU
     int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

     // for simplicity we are going to use square arrays
     nr_rows_A = 12;
     nr_cols_A = 500;
     nr_rows_B = 500;
     nr_cols_B = 90000;
     nr_rows_C = 12;
     nr_cols_C = 90000;

     // Allocate 3 arrays on GPU
     float *d_A, *d_B, *d_C;
     hipMalloc(&d_A,nr_rows_A * nr_cols_A * sizeof(float));
     hipMalloc(&d_B,nr_rows_B * nr_cols_B * sizeof(float));
     hipMalloc(&d_C,nr_rows_C * nr_cols_C * sizeof(float));

     // Fill the arrays A and B on GPU with random numbers
     GPU_fill_rand(d_A, nr_rows_A, nr_cols_A);
     GPU_fill_rand(d_B, nr_rows_B, nr_cols_B);

     // Multiply A and B on GPU
     gpu_blas_mmul(d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);

     // Copy (and print) the result on host memory

     //Free GPU memory
     hipFree(d_A);
     hipFree(d_B);
     hipFree(d_C);  

     return 0;
 }
